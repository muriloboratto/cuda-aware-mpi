/*%****************************************************************************80
%  Code: 
%   pingpong_CUDA-AWARE.c
%
%  Purpose:
%   Implements the ping pong between 2 ranks using CUDA-AWARE-MPI.
%   The code allocate memory between 2 GPUs using GPU direct.
%
%  Modified:
%   Dec 12 2021 10:57 
%
%  Author:
%   olcf-tutorials/MPI_ping_pong
%
%  Modified:
%   Murilo Boratto <murilo.boratto 'at' fieb.org.br>
%
%  HowtoCompile in nowherman:
%   
%
%  HowtoExecute in nowherman: 
%   
%
%****************************************************************************80*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <mpi.h>

int main(int argc, char *argv[]){

    int size, rank;

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    MPI_Status status;

    hipSetDevice(rank);

    double start_time, stop_time, elapsed_time;

    for(int i = 0; i <= 27; i++){

        long int N = 1 << i; /*Loop from 8 Bites to 1 GB*/
   
        double *A = (double*)calloc(N, sizeof(double)); /*Allocate memory for A on CPU*/

        double *d_A;

        hipMalloc(&d_A, N * sizeof(double)) ;
        hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice);

        int tag1 = 1000;
        int tag2 = 2000;

        int loop_count = 50;

       /********************************/      
       /**/ start_time = MPI_Wtime();/**/
       /********************************/

        for(int i = 1; i <= loop_count; i++){

            if(rank == 0){
                MPI_Send(d_A, N, MPI_DOUBLE, 1, tag1, MPI_COMM_WORLD);
                MPI_Recv(d_A, N, MPI_DOUBLE, 1, tag2, MPI_COMM_WORLD, &status);
            }
            else if(rank == 1){
                MPI_Recv(d_A, N, MPI_DOUBLE, 0, tag1, MPI_COMM_WORLD, &status);
                MPI_Send(d_A, N, MPI_DOUBLE, 0, tag2, MPI_COMM_WORLD);
            }
       
        }

       /**********************************/      
       /**/  stop_time = MPI_Wtime(); /**/
       /*********************************/

        /*measured*/
        elapsed_time = stop_time - start_time;
        long int num_B = 8 * N;
        long int B_in_GB = 1 << 30;
        double num_GB = (double)num_B / (double)B_in_GB;
        double avg_time_per_transfer = elapsed_time / (2.0*(double)loop_count);

        if(rank == 0) 
            printf("Transfer size (Bytes): %10li, Transfer Time (seconds): %15.9f, Bandwidth (GB/s): %15.9f\n", 
                    num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer );

        hipFree(d_A);
        free(A);
    
    }

    MPI_Finalize();

    return 0;

}/*main*/
