/*%****************************************************************************80
%  Code: 
%   ping-pong-MPI-CUDA.c
%
%  Purpose:
%   Implements the ping pong between 2 ranks using MPI and CUDA.
%   The code allocate memory between 2 GPUs.
%
%  Modified:
%   Dec 12 2021 10:57 
%
%  Author:
%   olcf-tutorials/MPI_ping_pong
%
%  Modified:
%   Murilo Boratto <murilo.boratto 'at' fieb.org.br>
%
%  HowtoCompile in nowherman:
%   nvcc -I/usr/include/openmpi -L/usr/lib/openmpi -lmpi -Xcompiler -fopenmp -o ping-pong-MPI-CUDA ping-pong-MPI-CUDA.cu
%
%  HowtoExecute in nowherman: 
%   mpirun -np 2 ./ping-pong-MPI-CUDA
%
%****************************************************************************80*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <mpi.h>

int main(int argc, char *argv[]){

    int size, rank;

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    MPI_Status status;

    hipSetDevice(rank);

    double start_time, stop_time, elapsed_time;

    for(int i = 0; i <= 27; i++){

        long int N = 1 << i; /*Loop from 8 Bites to 1 GB*/
   
        double *A = (double*)calloc(N, sizeof(double)); /*Allocate memory for A on CPU*/

        double *d_A;

        hipMalloc(&d_A, N * sizeof(double)) ;
        hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice);

        int tag1 = 1000;
        int tag2 = 2000;

        int loop_count = 50;

       /********************************/      
       /**/ start_time = MPI_Wtime();/**/
       /********************************/

        for(int i = 1; i <= loop_count; i++){

            if(rank == 0){
                hipMemcpy(A, d_A, N * sizeof(double), hipMemcpyDeviceToHost);
                MPI_Send(A, N, MPI_DOUBLE, 1, tag1, MPI_COMM_WORLD);
                MPI_Recv(A, N, MPI_DOUBLE, 1, tag2, MPI_COMM_WORLD, &status);
                hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice);
            }
            else if(rank == 1){
                MPI_Recv(A, N, MPI_DOUBLE, 0, tag1, MPI_COMM_WORLD, &status);
                hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice);
                hipMemcpy(A, d_A, N * sizeof(double), hipMemcpyDeviceToHost);
                MPI_Send(A, N, MPI_DOUBLE, 0, tag2, MPI_COMM_WORLD);
            }

        }/*for*/

       /**********************************/      
       /**/  stop_time = MPI_Wtime(); /**/
       /*********************************/

        /*measured*/
        elapsed_time = stop_time - start_time;
        long int num_B = 8 * N;
        long int B_in_GB = 1 << 30;
        double num_GB = (double)num_B / (double)B_in_GB;
        double avg_time_per_transfer = elapsed_time / (2.0*(double)loop_count);

        if(rank == 0) 
        	printf("Transfer size (Bites): %10li, Transfer Time (seconds): %15.9f, Bandwidth (GB/s): %15.9f\n", 
                    num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer );

        hipFree(d_A);
        free(A);
    
    }/*for*/

    MPI_Finalize();

    return 0;

}/*main*/
